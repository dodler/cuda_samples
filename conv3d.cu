#include "hip/hip_runtime.h"
/*
+ * conv_2d_cpu.cpp
 *
 *  Created on: Mar 5, 2018
 *      Author: lyan
 */
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/driver_types.h>

#include <ctime>
#include <iostream>
#include <cstdlib>

#include "common.h"
#include "gpu_conv_3d.h"

using namespace std;

/**
 * convolves in and kernel from top left corner defined by i j k tuple
 */
int convolve(tensor3 in, tensor3 kernel, int i,int j,int k, int kCols, int kRows, int kDepth){

	int result = 0;

	for(int m = 0; m<kCols; m++){
		for(int n = 0; n<kRows; n++){
			for(int p = 0; p<kDepth; p++){
				result += kernel[m][n][p] * in[i + m][j + n][k + p];
			}
		}
	}
	return result;
}


tensor3 conv_3d(tensor3 in, tensor3 kernel, int cols,int rows, int depth, int kCols,int kRows,int kd, int stride, int padding){

	if ((cols + 2 * padding - kCols) % stride != 0){
		cout << "bad stride" << endl;
	}

	if ((rows + 2 * padding - kRows) % stride != 0){
		cout << "bad stride" << endl;
	}

	if ((depth + 2 * padding - kd) % stride != 0){
		cout << "bad stride" << endl;
	}

	int rCols = (cols + 2 * padding - kCols) / stride + 1;
	int rRows = (rows + 2 * padding - kRows) / stride + 1;
	int rDepth = (depth + 2 * padding - kd) / stride + 1;

	tensor3 out = initVals(
			(cols + 2 * padding - kCols) / stride + 1,
			(rows + 2 * padding - kRows) / stride + 1,
			(depth + 2 * padding - kd) / stride + 1,
			0);

	tensor3 inPad = pad(in, cols, rows, depth, padding);

//	printSlice(inPad, 1, cols + 2 * padding, rows + 2 * padding);

	int m = 0, n = 0, l = 0;

	for(int i = 0; m<rCols; i+= stride, m++){
		for(int j = 0; n<rRows; j+= stride, n++){
			for(int k = 0; l<rDepth; k+=stride, l++){
				int t = convolve(inPad, kernel, i,j,k, kCols, kRows, kd);
				out[m][n][l] = t;
			}
			l=0;
		}
		n=0;
	}

	deleteTensor3(inPad, cols + 2 * padding, rows + 2 * padding, depth + 2 * padding);

	return out;
}

tensor3 cpu_test(tensor3 large, tensor3 kernel, int dim, int kdim, int padding, int stride, int resDim, int iter){

	cout << "starting cpu test" << endl;

	cout << "starting convolution" << endl;
	clock_t start;
	double total = 0;

	for(int i = 0; i<iter; i++){
		start = clock();
		tensor3 con = conv_3d(large, kernel, dim,dim,dim,kdim,kdim,kdim,stride,padding);
		total += double(clock() - start) / CLOCKS_PER_SEC;
		deleteTensor3(con, resDim,resDim,resDim);
	}

	cout << "avg time:" << total / ITER << endl;

	cout << "convolution done" << endl;

	tensor3 con = conv_3d(large, kernel, dim,dim,dim,kdim,kdim,kdim,stride,padding);
//	printSlice(con, 0, resDim,resDim);
//	cout << "-------------------------" << endl;
//	printSlice(con, 1, resDim,resDim);

	return con;
}



__host__ tensor3 gpu_test(tensor3 large, tensor3 kernel,int dim, int kdim, int padding, int stride, int resDim, int iter){
	return conv_3d_gpu(large, kernel, dim,dim,dim,kdim,kdim,kdim, padding, stride, iter);
}

bool test_conv(int dim, int kdim, int padding, int stride){
	int resDim = (dim + 2 * padding - kdim) / stride + 1;

	tensor3 large = initSeq(dim);
	tensor3 kernel = initVals(kdim,1);

	tensor3 res = cpu_test(large, kernel, dim,kdim,padding, stride, resDim, 1);
	tensor3 g_res = gpu_test(large, kernel, dim,kdim, padding,stride, resDim,1);

	bool eq = equal(res, g_res, resDim, resDim,resDim);

	deleteTensor3(res, resDim,resDim,resDim);
	deleteTensor3(g_res, resDim,resDim,resDim);

	deleteTensor3(large, dim, dim, dim);
	deleteTensor3(kernel, kdim, kdim, kdim);

	return eq;
}

__host__ int main(){
	int dim = 20;
	int kdim = 3;
	int padding = 0;
	int stride = 1;
	int resDim = (dim + 2 * padding - kdim) / stride + 1;

	cout << test_conv(2, 1, 0, 1) << endl;
	cout << test_conv(2, 1, 1, 1) << endl;
	cout << test_conv(3, 1, 1, 1) << endl;
	cout << test_conv(3, 2, 0, 1) << endl;
	cout << test_conv(9, 2, 1, 1) << endl;
	cout << test_conv(10, 2, 1, 1) << endl;
	cout << test_conv(17, 2, 0, 1) << endl;
	cout << test_conv(16, 2, 1, 1) << endl;
	cout << test_conv(28, 4, 2, 1) << endl;
	cout << test_conv(28, 4, 0, 1) << endl;
	cout << test_conv(50, 3, 0, 1) << endl;
	cout << test_conv(50, 3, 1, 1) << endl;
	cout << test_conv(100, 3, 0, 1) << endl;
	cout << test_conv(100, 3, 1, 1) << endl;
	cout << test_conv(200, 3, 1, 1) << endl;
	cout << test_conv(200, 3, 3, 1) << endl;
	cout << test_conv(200, 2, 5, 1) << endl;

}
