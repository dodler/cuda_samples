#include "hip/hip_runtime.h"
/*
+ * conv_2d_cpu.cpp
 *
 *  Created on: Mar 5, 2018
 *      Author: lyan
 */
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/driver_types.h>

#include <ctime>
#include <iostream>
#include <cstdlib>

#include "common.h"
#include "gpu_conv_3d.h"

using namespace std;

/**
 * convolves in and kernel from top left corner defined by i j k tuple
 */
int convolve(tensor3 in, tensor3 kernel, int i,int j,int k, int kCols, int kRows, int kDepth){

	int result = 0;

	for(int m = 0; m<kCols; m++){
		for(int n = 0; n<kRows; n++){
			for(int p = 0; p<kDepth; p++){
				result += kernel[m][n][p] * in[i + m][j + n][k + p];
			}
		}
	}
	return result;
}


tensor3 conv_3d(tensor3 in, tensor3 kernel, int cols,int rows, int depth, int kCols,int kRows,int kd, int stride, int padding){

	if ((cols + 2 * padding - kCols) % stride != 0){
		cout << "bad stride" << endl;
	}

	if ((rows + 2 * padding - kRows) % stride != 0){
		cout << "bad stride" << endl;
	}

	if ((depth + 2 * padding - kd) % stride != 0){
		cout << "bad stride" << endl;
	}

	int rCols = (cols + 2 * padding - kCols) / stride + 1;
	int rRows = (rows + 2 * padding - kRows) / stride + 1;
	int rDepth = (depth + 2 * padding - kd) / stride + 1;

//	cout << (cols + 2 * padding - kCols) / stride + 1 << endl;
//	cout << (rows + 2 * padding - kRows) / stride + 1 << endl;
//	cout << (depth + 2 * padding - kd) / stride + 1  << endl;

	tensor3 out = initVals(
			(cols + 2 * padding - kCols) / stride + 1,
			(rows + 2 * padding - kRows) / stride + 1,
			(depth + 2 * padding - kd) / stride + 1,
			0);

	tensor3 inPad = pad(in, cols, rows, depth, padding);

	int m = 0, n = 0, l = 0;

	for(int i = 0; m<rCols; i+= stride, m++){
		for(int j = 0; n<rRows; j+= stride, n++){
			for(int k = 0; l<rDepth; k+=stride, l++){
				int t = convolve(inPad, kernel, i,j,k, kCols, kRows, kd);
				out[m][n][l] = t;
			}
			l=0;
		}
		n=0;
	}

	deleteTensor3(inPad, cols + 2 * padding, rows + 2 * padding, depth + 2 * padding);

	return out;
}

void cpu_test(int dim, int kdim, int padding, int stride, int resDim){

	cout << "starting cpu test" << endl;

	tensor3 large = initSeq(dim);
	tensor3 kernel = initVals(kdim,2);

	cout << "starting convolution" << endl;
	clock_t start;
	double total = 0;
	int ITER = 2;
	for(int i = 0; i<ITER; i++){
		start = clock();
		tensor3 con = conv_3d(large, kernel, dim,dim,dim,kdim,kdim,kdim,1,0);
		total += double(clock() - start) / CLOCKS_PER_SEC;
		deleteTensor3(con, resDim,resDim,resDim);
	}

	cout << "avg time:" << total / ITER << endl;

	cout << "convolution done" << endl;

	deleteTensor3(large, dim, dim, dim);
	deleteTensor3(kernel, kdim, kdim, kdim);
}



__host__ void gpu_test(int dim, int kdim, int padding, int stride, int resDim){

	cout << "starting gpu test" << endl;

	tensor3 large = initSeq(dim);
	tensor3 kernel = initVals(kdim,2);

	cout << "copying data to gpu" << endl;

	int* g_large = initGpuTensor(large, dim, dim,dim);
	int* g_kernel = initGpuTensor(kernel, kdim,kdim,kdim);

	cout << "gpu ini donet" << endl;

	conv_3d_gpu(large, kernel, dim,dim,dim,kdim,kdim,kdim, padding, stride);

	cout << "conv done" << endl;

	int* out = new int[kdim * kdim * kdim];
	CUDA_CHECK_RETURN(hipMemcpy(out, g_kernel, t3_int_size(kdim,kdim,kdim), hipMemcpyDeviceToHost));
	cout << out[0] << " " << out[1] << endl;

	CUDA_CHECK_RETURN(hipFree(g_large));
	CUDA_CHECK_RETURN(hipFree(g_kernel));

	deleteTensor3(large, dim, dim, dim);
	deleteTensor3(kernel, kdim, kdim, kdim);

	cout << "gpu test end" << endl;
}

__host__ int main(){
	int dim = 81;
	int kdim = 3;
	int padding = 0;
	int stride = 1;
	int resDim = (dim + 2 * padding - kdim) / stride + 1;

	cpu_test(dim, kdim, padding, stride,resDim);
	gpu_test(dim,kdim, padding,stride, resDim);
}
